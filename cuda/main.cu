#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <float.h>
#include <cutil.h>

#ifdef FPS
#include <time.h>
#endif

#define pi      3.141592653589
#define BLACK   0
#define RED     1
#define EPSILON DBL_EPSILON

int keys[256];
int plot_sizex;  
int plot_sizey;
int win;

typedef unsigned long long int ullong;
#define FPS
#define RIC   0
#define ff    8 

#define N        (int)(512*ff*ff)
#define L        20.0
#define radius   (0.2*2.5/ff)
#define Npercell  36

#define epsilon  46.0
#define speed    0.2

#define vhappy_black  0.0
#define vhappy_red    0.2
#define damp_coeff    1.0

#define dt  1e-1
#define R   2*radius 
#define R2  R*R
#define FR  2*R
#define FR2 FR*FR

#define TILE    16 
#define TILEX   TILE
#define TILEY   TILE
#define THREADS ((int)(N/(TILE*TILE)))

#define ERROR_CHECK { hipError_t err; \
  if ((err = hipGetLastError()) != hipSuccess) { \
    printf("CUDA error: %s, line %d\n", hipGetErrorString(err), __LINE__);}}

//-----------------------------------------------------------
// some defines and what not 
//------------------------------------------------------------
void   ran_seed(long j);
float  ran_ran2();
ullong vseed;
ullong vran;

void simulate(float a, float e, int s);
void init_circle(float *x, float *v, int *t);

float eps = EPSILON; 

__device__ float mymod(float a, float b){
  return a - b*(int)(a/b) + b*(a<0);
}

__device__ void coords_to_index(float *x, int *size, int *index){   
    index[0] = (int)(x[0]/L  * size[0]);
    index[1] = (int)(x[1]/L  * size[1]);
}

__device__ int mod_rvec(int a, int b, int *image){
    *image = 1;
    if (b==0) {if (a==0) *image=0; return 0;}
    if (a>b)  return a-b-1;
    if (a<0)  return a+b+1;
    *image = 0;
    return a;
}

void coords_to_index2(float *x, int *size, int *index){   
    index[0] = (int)(x[0]/L  * size[0]);
    index[1] = (int)(x[1]/L  * size[1]);
}

//===================================================
// the main function
//===================================================
int main(int argc, char **argv){
    float alpha_in = 0.1; 
    float eta_in   = 0.1;
    int seed_in     = 0;
    
    CUT_DEVICE_INIT(argc, argv);

    //if (argc == 1) 
        simulate(alpha_in, eta_in, seed_in);
    /*else if (argc == 4){
        alpha_in = atof(argv[1]);
        eta_in   = atof(argv[2]);
        seed_in  = atoi(argv[3]);
        simulate(alpha_in, eta_in, seed_in);
    }
    else {
        printf("usage:\n");
        printf("\t./entbody [alpha] [eta] [seed]\n");
    }*/
    return 0;
}


__global__ void step(float *x, float *v, int *type, 
                     unsigned int *cells, unsigned int *count, 
                     float *col, int *size, int size_total){
    //int bx = blockIdx.x;     
    //int by = blockIdx.y;     
    int tx = threadIdx.x;    

    //int dx = blockDim.x;
    //int dy = blockDim.y;
 
    int n = blockDim.x*blockIdx.x + tx;

    int index[2];
    if (n < size_total)
        count[n] = 0;
    if (n < size_total*Npercell)
        cells[n] = 0;

    __syncthreads();

    coords_to_index(&x[2*n], size, index);
    int t = index[0] + index[1]*size[0];
    unsigned int pos = atomicInc(&count[t], 0xffffffff);
    cells[Npercell*t + pos] = n;

    __syncthreads();

    int j,k;
    int tt[2];
    int tix[2];
    int image[2];
    float dx[2];

    float fx = 0.0;
    float fy = 0.0;
    float wx = 0.0;
    float wy = 0.0;
    
    for (tt[0]=-1; tt[0]<=1; tt[0]++){
    for (tt[1]=-1; tt[1]<=1; tt[1]++){
        tix[0] = mod_rvec(index[0]+tt[0],size[0]-1,&image[0]);
        tix[1] = mod_rvec(index[1]+tt[1],size[1]-1,&image[1]);

        int ind = tix[0] + tix[1]*size[0]; 

        for (j=0; j<count[ind]; j++){
            int tn = cells[Npercell*ind+j];

            float dist = 0.0;
            for (k=0; k<2; k++){
                dx[k] = x[2*tn+k] - x[2*n+k];
        
                if (image[k])
                    dx[k] += L*tt[k];
                dist += dx[k]*dx[k];
            }

            //===============================================
            // force calculation - hertz
            if (dist > 1e-6 && dist < R2){
                float r0 = R; 
                float l  = sqrt(dist);
                float co = epsilon * (1-l/r0)*(1-l/r0) * (l<r0);
                fx += - dx[k] * co;
                fy += - dx[k] * co;
                col[n] += co*co*dx[0]*dx[0]; 
                col[n] += co*co*dx[1]*dx[1]; 
            }
            //===============================================
            // add up the neighbor veocities
            if (dist > 1e-6 && dist < FR2 && type[n] == RED && type[tn] == RED){
                wx += v[2*n+0];
                wy += v[2*n+1];
            }                           
        }
    } } 

    //=====================================
    // flocking force 
    float wlen = wx*wx + wy*wy;
    if (type[n] == RED && wlen > 1e-6){
        fx += speed * wx / wlen; 
        fy += speed * wy / wlen;
    }

    //====================================
    // self-propulsion
    float vlen = v[2*n+0]*v[2*n+0] + v[2*n+1]*v[2*n+1];
    float vhappy = type[n]==RED?vhappy_red:vhappy_black;
    if (vlen > 1e-6){
        fx += damp_coeff*(vhappy - vlen)*v[2*n+0]/vlen;
        fy += damp_coeff*(vhappy - vlen)*v[2*n+1]/vlen;
    }
    
    // Newton-Stomer-Verlet
    v[2*n+0] += fx * dt;
    v[2*n+1] += fy * dt;

    x[2*n+0] += v[2*n+0] * dt;
    x[2*n+1] += v[2*n+1] * dt;
    
    // boundary conditions 
    if (x[2*n+0] >= L-EPSILON || x[2*n+0] < 0)
        x[2*n+0] = mymod(x[2*n+0], L);
    if (x[2*n+1] >= L-EPSILON || x[2*n+1] < 0)
        x[2*n+1] = mymod(x[2*n+1], L);

    col[n] = col[n]/4; 
}

//==================================================
// simulation
//==================================================
void simulate(float alpha, float eta, int seed){
    printf("Simulating %i particles\n", N);

    ran_seed(seed);
    int i;
    //int exit = 0;

    printf("Freeing local memory...\n");
    int *type   = (int*)malloc(sizeof(int)*N);
    float *rad = (float*)malloc(sizeof(float)*N); 
    float *col = (float*)malloc(sizeof(float)*N); 
    for (i=0; i<N; i++){ type[i] = 0; rad[i] = 0.0;}

    float *x = (float*)malloc(sizeof(float)*2*N);
    float *v = (float*)malloc(sizeof(float)*2*N);
    float *f = (float*)malloc(sizeof(float)*2*N);
    float *w = (float*)malloc(sizeof(float)*2*N);
    for (i=0; i<2*N; i++){x[i] = v[i] = f[i] = w[i] = 0.0;}

    float time_end = 1e2;

    #ifdef PLOT
    plot_init(); 
    plot_clear_screen();
    #endif

    //-------------------------------------------------
    // initialize
    printf("Initializing...\n");
    if (RIC){
        for (i=0; i<N; i++){
            float t = 2*pi*ran_ran2();
    
            rad[i] = radius;
            x[2*i+0] = L*ran_ran2();
            x[2*i+1] = L*ran_ran2();
     
            if (ran_ran2() > 0.3){
                v[2*i+0] = 0.0;
                v[2*i+1] = 0.0;
                type[i] = BLACK;
            }
            else {
                v[2*i+0] = vhappy_red * sin(t);
                v[2*i+1] = vhappy_red * cos(t);
                type[i] = RED;
            } 
        }
    }
    else {
        for (i=0; i<N; i++)
            rad[i] = radius;
        init_circle(x, v, type);
    }

    //-------------------------------------------------------
    // make boxes for the neighborlist
    int size[2];
    int size_total = 1;
    for (i=0; i<2; i++){
        size[i] = (int)(L / (FR)); 
        size_total *= size[i];
    }

    unsigned int *count  = (unsigned int*)malloc(sizeof(unsigned int)*size_total);
    unsigned int *cells  = (unsigned int*)malloc(sizeof(unsigned int*)*Npercell*size_total);
    unsigned int *count2  = (unsigned int*)malloc(sizeof(unsigned int)*size_total);
    unsigned int *cells2  = (unsigned int*)malloc(sizeof(unsigned int*)*Npercell*size_total);
    for (i=0; i<size_total; i++)
        count[i] = 0;
    for (i=0; i<size_total*Npercell; i++)
        cells[i] = 0;

    int index[2];
    for (i=0; i<N; i++){
        coords_to_index2(&x[2*i], size, index);
        int t = index[0] + index[1]*size[0];
        cells[Npercell*t + count[t]] = i;
        count[t]++; 
    }


    //==========================================================
    // where the magic happens
    //==========================================================
    printf("Freeing device memory...\n");
    int mem_size2 = sizeof(int)*2;
    int imem_size = sizeof(int)*N;
    int fmem_size = sizeof(float)*N;
    int fmem_siz2 = sizeof(float)*N*2;
    int mem_cell  = sizeof(unsigned int)*size_total;
    int mem_cell2 = sizeof(unsigned int)*size_total*Npercell;

    unsigned int *cu_count  = NULL;
    unsigned int *cu_cells  = NULL;    
    int *cu_size   = NULL;
    int *cu_type   = NULL; 

    float *cu_rad  = NULL;
    float *cu_col  = NULL;
    float *cu_x    = NULL;
    float *cu_v    = NULL; 
 
    hipMalloc((void**) &cu_count, mem_cell);
    hipMalloc((void**) &cu_cells, mem_cell2);
    hipMalloc((void**) &cu_size,  mem_size2); 
 
    hipMalloc((void**) &cu_type,  imem_size);
    hipMalloc((void**) &cu_rad,   fmem_size);
    hipMalloc((void**) &cu_col,   fmem_size);
    hipMalloc((void**) &cu_x,     fmem_siz2);
    hipMalloc((void**) &cu_v,     fmem_siz2);
    
    printf("Copying problem...\n");
    hipMemcpy(cu_size,  size,  mem_size2, hipMemcpyHostToDevice);
    hipMemcpy(cu_type,  type,  imem_size, hipMemcpyHostToDevice);
    hipMemcpy(cu_rad,   rad,   fmem_size, hipMemcpyHostToDevice);
    hipMemcpy(cu_col,   col,   fmem_size, hipMemcpyHostToDevice);
    hipMemcpy(cu_x,     x,     fmem_siz2, hipMemcpyHostToDevice);
    hipMemcpy(cu_v,     v,     fmem_siz2, hipMemcpyHostToDevice);
    
    hipMemset(cu_count, 0, mem_cell);
    hipMemset(cu_cells, 0, mem_cell2);
    ERROR_CHECK

    // ================================================
    // Initialize the block and grid dimensions here
    // ================================================
    #ifdef FPS
    struct timespec start;
    clock_gettime(CLOCK_REALTIME, &start);
    #endif

    int frames = 0;
    float t=0.0;

    dim3 grid(TILEX, TILEY, 1);
    dim3 thrd(THREADS, 1, 1);
    
    struct timespec startt;
    clock_gettime(CLOCK_REALTIME, &startt);

    int blocks = 256;
    printf("Simulating %ix%i = %i...\n", blocks, N/blocks);
    for (t=0.0; t<time_end; t+=dt){
 
        //step<<<grid, thrd>>>(cu_x, cu_v, cu_type, cu_cells, cu_count, cu_col, cu_size, size_total);
        step<<<blocks,N/blocks>>>(cu_x, cu_v, cu_type, cu_cells, cu_count, cu_col, cu_size, size_total);
        hipDeviceSynchronize();

        frames++;
        if (frames % 100 == 0){
            struct timespec end;
            clock_gettime(CLOCK_REALTIME, &end);
            printf("%i : %f\n", frames, (100)/((end.tv_sec - startt.tv_sec) + (end.tv_nsec - startt.tv_nsec)/1e9));
            ERROR_CHECK
            clock_gettime(CLOCK_REALTIME, &startt);
        }

        #ifdef PLOT
        plot_clear_screen();
        plot_render_particles(x,col,etc  );
        #endif
    }

    #ifdef FPS
    struct timespec end;
    clock_gettime(CLOCK_REALTIME, &end);
    printf("frames = %i\n", frames);
    printf("time   = %f\n",(end.tv_sec - start.tv_sec) + (end.tv_nsec - start.tv_nsec)/1e9 );
    printf("fps = %f\n", frames/((end.tv_sec - start.tv_sec) + (end.tv_nsec - start.tv_nsec)/1e9));
    #endif

    hipMemcpy(cells2, cu_cells, mem_cell2, hipMemcpyDeviceToHost);
    hipMemcpy(count2, cu_count, mem_cell, hipMemcpyDeviceToHost);
    hipMemcpy(type, cu_type, imem_size, hipMemcpyDeviceToHost);

    int tcc = 0;
    int tcc2 = 0;
    for (i=0; i<size_total; i++){
        tcc += count[i];
        tcc2 += count2[i];
    }
    printf("-- %i %i --\n", tcc, tcc2);

    hipFree(cu_count);
    hipFree(cu_cells);
    hipFree(cu_type);
    hipFree(cu_rad);
    hipFree(cu_col);
    hipFree(cu_x);
    hipFree(cu_v);
    ERROR_CHECK
  
    free(cells);
    free(count);
 
    free(x);
    free(v);
    free(f);
    free(w);
    free(rad);
    free(type);

    #ifdef PLOT
    plot_clean(); 
    #endif
}




//=================================================
// extra stuff
//=================================================
void ran_seed(long j){
  vseed = j;  vran = 4101842887655102017LL;
  vran ^= vseed; 
  vran ^= vran >> 21; vran ^= vran << 35; vran ^= vran >> 4;
  vran = vran * 2685821657736338717LL;
}

float ran_ran2(){
    vran ^= vran >> 21; vran ^= vran << 35; vran ^= vran >> 4;
    ullong t = vran * 2685821657736338717LL;
    return 5.42101086242752217e-20*t;
}

void init_circle(float *x, float *v, int *type){
    int i;
    for (i=0; i<N; i++){
        float tx = L*ran_ran2();
        float ty = L*ran_ran2();
        float tt = 2*pi*ran_ran2();

        x[2*i+0] = tx;
        x[2*i+1] = ty;
        
        float dd = sqrt((tx-L/2)*(tx-L/2) + (ty-L/2)*(ty-L/2));

        // the radius for which 30% of the particles are red on avg
        float rad = sqrt(0.15*L*L / pi);
        if (dd < rad)
            type[i] = RED;

        if (type[i] == RED){
            v[2*i+0] = vhappy_red*cos(tt);
            v[2*i+1] = vhappy_red*sin(tt);
        }
        else {
            v[2*i+0] = 0.0;
            v[2*i+1] = 0.0;
        }
    }   
} 


#ifdef PLOT
//========================================================
// all of the plotting functionality
//========================================================
void key_down(unsigned char key, int x, int y){
    keys[key] = 1;
}
void key_up(unsigned char key, int x, int y){
    keys[key] = 0;
}

void plot_init(){
  plot_sizex = 640;
  plot_sizey = 640;
  win = 0;
  plot_init_opengl();
    int i;
    for (i=0; i<256; i++)
        keys[i] = 0;
}

void plot_clean(){
  plot_end_opengl();
}

//=============================================================
// OpenGL functionality
// http://www.andyofniall.net/2d-graphics-with-opengl/
//=============================================================
void plot_init_opengl(){
  int argc = 1;
  char *argv = (char*)malloc(sizeof(char)*42);
  sprintf(argv, "./entbody");

  glutInit(&argc, &argv);	         
  glutInitDisplayMode(GLUT_RGBA | GLUT_DOUBLE);
  glutInitWindowSize(plot_sizex, plot_sizey);
  glutInitWindowPosition(100,100);
  win = glutCreateWindow("EntBody Simulation");	

  glDisable(GL_DEPTH_TEST);
  glClearColor(1.0, 1.0, 1.0, 0.0);	/* set background to white */
  glutKeyboardFunc(key_down);
  glutKeyboardUpFunc(key_up);
  glViewport(0,0,plot_sizex, plot_sizey);

  glutMainLoopEvent();
  free(argv);
}

void plot_end_opengl(){
  glutDestroyWindow(win);
}

int plot_clear_screen(){
  glClear(GL_COLOR_BUFFER_BIT);
  return 1;
}


int plot_render_particles(double *x, double *rad, int *type, double *shade){
    // focus on the part of scene where we draw nice
    glMatrixMode(GL_PROJECTION);
    glLoadIdentity();
    glOrtho(0, L, L, 0, 0, 1);
    glMatrixMode(GL_MODELVIEW);
    glLoadIdentity();

    // lets draw our viewport just in case its not square
    glBegin(GL_LINE_LOOP);
      glVertex2f(0, 0);
      glVertex2f(0, L);
      glVertex2f(L, L);
      glVertex2f(L, 0);
    glEnd();
    
    glDisable(GL_POINT_SMOOTH);
    glPointSize(3);

    #ifdef POINTS 
    glBegin(GL_POINTS);
    #else
    double t=0;
    #endif

    int i; 
    for (i=0; i<N; i++){
        float tx = (float)x[2*i+0];
        float ty = (float)x[2*i+1];

        double c = fabs(shade[i]);
        if (c < 0) c = 0.0;
        if (c > 1.0) c = 1.0;

        float cr = c;
        float cg = c;
        float cb = c;
        float ca = 1.0;

        if (type[i] == 1) {
            cr = 0.9;//if (cr < 0.2) cr = 0.2;
            cg = 0.05;
            cb = 0.05;
        }
        
        #ifdef POINTS
        plot_set_draw_color(cr,cg,cb,ca);
        glVertex2f(tx, ty);
        #else
        double rx = rad[i];
        uint secs = 15;
        plot_set_draw_color(cr,cg,cb,ca);
        glBegin(GL_POLYGON);
        for (t=0; t<2*pi; t+=2*pi/secs)
          glVertex2f(tx + rx*cos(t), ty + rx*sin(t));
        glEnd();
        plot_set_draw_color(0.0,0.0,0.0,1.0);
        glBegin(GL_LINE_LOOP);
        for (t=0; t<2*pi; t+=2*pi/secs)
          glVertex2f(tx + rx*cos(t), ty + rx*sin(t));
        glEnd();
        #endif
    }
    #ifdef POINTS
    glEnd();
    #endif

    glutSwapBuffers();
    glutMainLoopEvent();

   return 0;
}

void plot_set_draw_color(float cr, float cg, float cb, float ca){
  glColor4f(cr, cg, cb, ca);
}
#endif
